#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <cstdio>

#include "algebra.cuh"
#include "object.cuh"

Object object_new(const int shape_type, const Shape* shape, const Float3* color,
				  const float emission_intensity, const float reflection) {
	static int id = 0;
	Object object;
	object.id = id++;
	object.shape = *shape;
	object.color = *color;
	object.light_emitted = float3_mul(color, emission_intensity);
	object.reflection = reflection;
	object.shape_type = shape_type;
	return object;
}

__device__ __host__ float object_intersect_distance(const Object* object,
													const Ray3* ray) {
	// switch (object->shape_type) {
	// 	case TYPE_SPHERE:
	// 		return sphere_intersect_distance(&object->shape, ray);
	// 	case TYPE_PLANE:
	// 		return plane_intersect_distance(&object->shape, ray);
	// 	case TYPE_TRIANGLE:
	// 		return triangle_intersect_distance(&object->shape, ray);
	// }
	// return -1;

	if (object->shape_type == TYPE_SPHERE) {
		return sphere_intersect_distance(&object->shape, ray);
	} else if (object->shape_type == TYPE_PLANE) {
		return plane_intersect_distance(&object->shape, ray);
	} else /* if (object->shape_type == TYPE_TRIANGLE)  */ {
		return triangle_intersect_distance(&object->shape, ray);
	}
}

__device__ __host__ Float3 object_normal_normalized(const Object* object,
													const Ray3* ray) {
	Float3 direction;

	// switch (object->shape_type) {
	// 	case TYPE_SPHERE:
	// 		direction = sphere_normal_normalized(&object->shape, &ray->origin);
	// 	case TYPE_PLANE:
	// 		direction = plane_normal_normalized(&object->shape, &ray->origin);
	// 	case TYPE_TRIANGLE:
	// 		direction =
	// 			triangle_normal_normalized(&object->shape, &ray->origin);
	// }

	if (object->shape_type == TYPE_SPHERE)
		direction = sphere_normal_normalized(&object->shape, &ray->origin);
	else if (object->shape_type == TYPE_PLANE)
		direction = plane_normal_normalized(&object->shape, &ray->origin);
	else /* if (object->shape_type == TYPE_TRIANGLE) */
		direction = triangle_normal_normalized(&object->shape, &ray->origin);

	if (float3_dot(&ray->direction, &direction) > 0.0)
		float3_invert_eq(&direction);
	return direction;
}

ObjectVec objectvec_new(const int n) {
	ObjectVec obj_container;
	obj_container.size = 0;
	obj_container.capacity = n;
	obj_container.ptr = (Object*)malloc(sizeof(Object) * n);
	if (obj_container.ptr == NULL) {
		fprintf(stderr, "Error: malloc failed in new_object_container()\n");
		exit(-1);
	}
	return obj_container;
}

void object_vec_push(ObjectVec* object_v, const Object* object) {
	if (object_v->size >= object_v->capacity) {
		fprintf(stderr, "Error: object_v->size >= object_v->capacity\n");
		exit(-1);
	}
	memcpy(object_v->ptr + object_v->size, object, sizeof(Object));
	object_v->size++;
}

__device__ void object_reflect_ray(const Object* object, Ray3* ray,
								   const float distance, hiprandState* state) {
	ray3_move_along(ray, distance);
	const Float3 normal = object_normal_normalized(object, ray);
	const float flip = (float)hiprand(state) / 4294967295.0f;
	if (flip < object->reflection) {
		ray->direction = float3_mirror(&ray->direction, &normal);
	} else {
		ray->direction = half_sphere_random(&normal, state);
	}
}

#define PI 3.14159265358979323846

__device__ Float3 half_sphere_random(const Float3* normal, hiprandState* state) {
	const float phi = 2 * PI * (float)hiprand(state) / 4294967295.0f;
	const float theta = PI * (float)hiprand(state) / 4294967295.0f;
	const float sin_theta = sinf(theta);
	Float3 retval =
		float3_new(sin_theta * cosf(phi), sin_theta * sinf(phi), cosf(theta));
	if (float3_dot(&retval, normal) < 0) {
		float3_invert_eq(&retval);
	}
	return retval;
}
